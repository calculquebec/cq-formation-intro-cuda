
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void setData(int *ptr) {
    *ptr = 7;
}


int main(int, char**) {
    int *data_d = 0;
    int *data_h = 0;

    hipError_t err;

    if ((err = hipMalloc((void**)&data_d, sizeof(int))) != hipSuccess) {
        printf("Could not allocate that much memory. \n%s",hipGetErrorString(err));
        exit(1);
    }
    data_h = (int *)malloc(sizeof(int));

    setData<<<1,1>>>(0);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error calling setData. \n%s",hipGetErrorString(err));
        goto cleanup;
    }

    err = hipMemcpy(data_h, data_d, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Could not copy memory \n%s",hipGetErrorString(err));
        goto cleanup;
    }

    printf("data = %d\n", *data_h);
    free(data_h);

cleanup:
    if ((err = hipFree(data_d)) != hipSuccess) {
        printf("Could not free memory (free #1) \n%s",hipGetErrorString(err));
        exit(1);
    }

    if ((err = hipFree(data_d)) != hipSuccess) {
        printf("Could not free memory (free #2) \n%s",hipGetErrorString(err));
        exit(1);
    }

    return 0;
}
