#include <stdio.h>
#include <assert.h>

void checkCUDAError(const char *msg);


int main(){

int count,size;

float *da;
hipDeviceProp_t *pDeviceProp;
	hipGetDeviceCount(&count);
	printf("count=%d\n",count);

	size=20*sizeof(float);
	hipMalloc((void**)&da,size);
	checkCUDAError("hipMemcpy calls");
	pDeviceProp = (hipDeviceProp_t*) malloc(sizeof(hipDeviceProp_t));

	hipSetDevice(6);
        checkCUDAError("Error setting a device\n");
	hipGetDeviceProperties(pDeviceProp,6);

	printf( "Device Name \t – %s ", pDeviceProp->name );
	printf( "\n**************************************");
	printf( "\nTotal Global Memory\t\t -%d KB", pDeviceProp->totalGlobalMem/1024 );
	printf( "\nShared memory available per block \t – %d KB", pDeviceProp->sharedMemPerBlock/1024 );
	printf( "\nNumber of registers per thread block \t – %d", pDeviceProp->regsPerBlock );
	printf( "\nWarp size in threads \t – %d", pDeviceProp->warpSize );
	printf( "\nMemory Pitch \t – %d bytes", pDeviceProp->memPitch );
	printf( "\nMaximum threads per block \t – %d", pDeviceProp->maxThreadsPerBlock );
	printf( "\nMaximum Thread Dimension (block) \t – %d %d %d", pDeviceProp->maxThreadsDim[0], pDeviceProp->maxThreadsDim[1], pDeviceProp->maxThreadsDim[2] );
	printf( "\nMaximum Thread Dimension (grid) \t – %d %d %d", pDeviceProp->maxGridSize[0], pDeviceProp->maxGridSize[1], pDeviceProp->maxGridSize[2] );
	printf( "\nTotal constant memory \t – %d bytes", pDeviceProp->totalConstMem );
	printf( "\nCUDA ver \t – %d.%d", pDeviceProp->major, pDeviceProp->minor );
	printf( "\nClock rate \t – %d KHz", pDeviceProp->clockRate );
	printf( "\nTexture Alignment \t – %d bytes", pDeviceProp->textureAlignment );
//	printf( "\nDevice Overlap \t – %s", pDeviceProp-> deviceOverlap);
	printf( "\nNumber of Multi processors \t – %d", pDeviceProp->multiProcessorCount );


}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }
}
