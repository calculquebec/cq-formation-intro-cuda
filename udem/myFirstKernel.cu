#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <assert.h>

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

// Part 3 of 5: implement the kernel
__global__ void myFirstKernel(int *d_a)
{
	// We want each thread do the job, so we need
	// a global thread index to access elements of the array


	// Make each element of the array equal .e.g. 1000

}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    // pointer for host memory
    int *h_a;

    // pointer for device memory
    int *d_a;

    // define grid and block size
    int numBlocks = 8;
    int numThreadsPerBlock = 8;

    // Set the device to be used
    // Put the DeviceId assigned to you
    int DeviceId=0;
    hipSetDevice(DeviceId);

    // Part 1 of 5: allocate host and device memory
    // Size of the array should be equal to the total number of threads
    size_t memSize = ;
    h_a = ;
    hipMalloc();

    // Part 2 of 5: configure and launch kernel
    // Set up one dimensional grid and thread-block
    dim3 dimGrid();
    dim3 dimBlock();
    myFirstKernel<<<   >>>(d_a);

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    checkCUDAError("kernel execution");

    // Part 4 of 5: device to host copy
    // Copy d_a to h_a
    hipMemcpy();

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy");

    // Part 5 of 5: verify the data returned to the host is correct
    // Use "assert" command to check that each element of the array equal to 1000 

    // free device memory
    hipFree(d_a);

    // free host memory
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }                         
}
