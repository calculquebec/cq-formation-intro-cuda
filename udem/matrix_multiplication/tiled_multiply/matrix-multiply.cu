#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <assert.h>
#include <time.h>
#include <sys/time.h>

#define WIDTH 4096
#define TILE_WIDTH 16

void checkCUDAError(const char *msg);
void fillRandomSingle(int m, int n, float* a, float min, float max);
double getHighResolutionTime(void);

__global__ void MatrixMultKernel(float *Md, float *Nd, float *Pd, int Width)
{

	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
	float Pvalue=0;
  	for(int k=0; k< Width; k++){
		Pvalue += Md[Row*Width + k] * Nd[k*Width + Col];
	}
	Pd[Row*Width + Col] = Pvalue;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
	int DeviceId=6;
	hipSetDevice(DeviceId);

	int Width, i,j,k;
	float *M;
	float *N;
	float *P;
	float *Md;
	float *Nd;
	float *Pd;

	Width=WIDTH;
	int size=Width*Width*sizeof(float);

	M = (float*)malloc(size);
	N = (float*)malloc(size);

	hipMalloc((void**)&Md, size);
	hipMalloc((void**)&Nd, size);

	fillRandomSingle(Width, Width, M, -10.0, 10.0);
	fillRandomSingle(Width, Width, N, -10.0, 10.0);
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
	checkCUDAError("Failed to copy data to GPU");

	// Allocate P on the host and device
	P = (float*)malloc(size);
	hipMalloc((void**)&Pd, size);

	printf("Width=%d\n",Width);
	// Setup the execution kernel grid
	dim3 dimGrid(Width/TILE_WIDTH, Width/TILE_WIDTH);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

	// Launch kernel
	double start_time = getHighResolutionTime();
	MatrixMultKernel<<< dimGrid, dimBlock >>> (Md, Nd, Pd, Width);
	hipDeviceSynchronize();
	double end_time = getHighResolutionTime();
	printf("Exec.time=%f\n",end_time-start_time);
	checkCUDAError("Kernel failed");

	// Read Pd from the device 
	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);

	// Free device matrices
	hipFree(Md); hipFree(Nd); hipFree(Pd);

	return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }                         
}

void fillRandomSingle(int m, int n, float* a, float min, float max)
{
    int i, j;

    srand(1);

    for (j=0; j<m; j++)
    {
        for (i=0; i<n; i++)
        {
            a[j*n+i] = min + (max-min) * rand()/RAND_MAX;
        }
    }
}

double getHighResolutionTime(void)
{
    struct timeval tod;

    gettimeofday(&tod, NULL);
    double time_seconds = (double) tod.tv_sec + ((double) tod.tv_usec / 1000000.0);
    return time_seconds;
}
