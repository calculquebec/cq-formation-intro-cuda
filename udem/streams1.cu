#include "hip/hip_runtime.h"
// In this example a HOST array a[] is copied to Device array d_a, then Kernel is launched
// The Kernel fill up the d_a[] array with some numbers. Then result is copied back to Host array a[]
// Make these 3 operations concurrent, i.e. run them in 4 different streams.


// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

# define SIZE 4096

// STEP 4 of 8: Implement a KERNEL
// Write a CUDA kernel that fill up an array with some integers numbers,
// make the kernel more time consuming by including a loop that just burns time 
__global__ void kernel(int *d_a,int offset){
	// Make all the threads and blocks work. Create a "global" thread index 
	int idx = ;
	
	// Put data in the array
	d_a[]=idx*2;
}

void checkCUDAError(const char *msg);
int main(int argc, char **argv)
{
    	int ndevices;
	int cuda_device = 1;
    	int nstreams = 4;
	int streamSize;	

	int *a;
	int *d_a;

	// Count number of GPUs on board
	hipGetDeviceCount(&ndevices);
	checkCUDAError("hipGetDevice failed !");
	printf("Number of GPUs available to run = %d\n",ndevices);

	// Set the GPU device
        hipSetDevice(0);
	checkCUDAError("hipSetDevice failed !");


	// Check CUDA Device properties for whether overlap between kernel & memcpy is supported
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, cuda_device);
	printf("Device: <%s> canMapHostMemory: %s\n", deviceProp.name, deviceProp.canMapHostMemory ? "Yes" : "No");
	printf("Number of copy engines = %d\n",deviceProp.asyncEngineCount);

	// STEP 1 of 8: Allocate memory of GPU
	// CUDA memory allocation




	// STEP 2 of 8: Allocate pinned memory on HOST
	// Paged-locked memory allocation of HOST




	//STEP 3 of 8: Create CUDA streams
    	hipStream_t *streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));
    	for(int i=0;i<nstreams;i++) {
	

	}

	//generate data
	for(int i=0;i<SIZE;i++) a[i]=0;

	// STEP 5 of 8: Create CUDA grid (blocks, threads) in a such a way that the arrays a[] or d_a[]
	// are handled in chunks. Number of chunks = number of streams, so each stream makes the copies 
	// and perform kernel operations only on its own chunk.
	streamSize = SIZE/nstreams;




	// STEP 6 of 8: Main LOOP that should include D2H, H2D copies, and a Kernel invocation





	// STEP 7 of 8: Synchronize streams
	for(int i=0;i<nstreams;i++) {
	
	}



	// STEP 8 of 8: Destroy streams
	for(int i=0;i<nstreams;i++) {
	}

}


void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }
}
