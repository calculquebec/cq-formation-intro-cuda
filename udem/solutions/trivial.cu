// includes, system
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

int main( int argc, char** argv) 
{
    int DeviceId=1;
    int numdevices;

    // STEP 1: Get number of devices available
    hipGetDeviceCount(&numdevices);

    // STEP 2: Set the Device you will work with
    hipSetDevice(DeviceId);

    // STEP 3: Get the last error message printed out
    printf("Checking last error: %s\n",hipGetErrorString(hipGetLastError()));

}

