#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <assert.h>

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

// Part 5: implement the kernel
__global__ void reverseArrayBlock(int *d_out, int *d_in)
{
    // create array original and reverse indices
    int in = threadIdx.x;
    int out = blockDim.x - 1 - threadIdx.x;

    // reverse the array content using appropriate indices
    d_out[out] = d_in[in];
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    // pointer for host memory and size
    int *h_a;
    int dimA = 256;

    // pointer for device memory
    int *d_b, *d_a;

    // define grid and block size
    int numBlocks = 1;
    int numThreadsPerBlock = dimA;

    // Set the device to be used
    // Put the DeviceId assigned to you
    int DeviceId=0;
    hipSetDevice(DeviceId);

    // Part 1: allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int *) malloc(memSize);
    hipMalloc( (void **) &d_a, memSize );
    hipMalloc( (void **) &d_b, memSize );

    // Part 2: Initialize input array on host
    for (int i = 0; i < dimA; ++i)
    {
        h_a[i] = i;
    }

    // Part 3: Copy host array to device array
    hipMemcpy( d_a, h_a, memSize, hipMemcpyHostToDevice );

    // Part 4: launch kernel
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
    reverseArrayBlock<<< dimGrid, dimBlock >>>( d_b, d_a );

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");

    // Part 6: device to host copy
    hipMemcpy( h_a, d_b, memSize, hipMemcpyDeviceToHost );

    // Check for any CUDA errors
    checkCUDAError("memcpy");

    // Part 7: verify the data returned to the host is correct
    for (int i = 0; i < dimA; i++)
    {
        assert(h_a[i] == dimA - 1 - i );
    }

    // free device memory
    hipFree(d_a);
    hipFree(d_b);

    // free host memory
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}
