#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <assert.h>

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

// Part 6: implement the kernel
__global__ void reverseArrayBlock( int* d_out, int* d_in )
{

    // create original and reverse array indices
    // keeping in mind that you have multiple blocks reversing the array content
    int in = blockIdx.x * blockDim.x + threadIdx.x;
    int out = gridDim.x * blockDim.x - in - 1;

    // reverse array using appropriate indices
    d_out[out] = d_in[in];
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    // pointer for host memory and size
    int *h_a;
    int dimA = 256 * 1024; // 256K elements (1MB total)

    // pointer for device memory
    int *d_b, *d_a;

    // define grid and block size
    int numThreadsPerBlock = 256;

    // Part 1: compute number of blocks needed based on array size and desired block size
    int numBlocks = dimA / numThreadsPerBlock;  

    // Part 2: allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int *) malloc(memSize);
    hipMalloc( (void **) &d_a, memSize );
    hipMalloc( (void **) &d_b, memSize );

    // Part 3: Initialize input array on host
    for (int i = 0; i < dimA; ++i)
    {
        h_a[i] = i;
    }

    // Part 4: Copy host array to device array
    hipMemcpy( d_a, h_a, memSize, hipMemcpyHostToDevice );

    // Part 5: Set up grid and launch kernel
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
    reverseArrayBlock<<< dimGrid, dimBlock >>>( d_b, d_a );

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");

    // Part 7: device to host copy
    hipMemcpy( h_a, d_b, memSize, hipMemcpyDeviceToHost );

    // Check for any CUDA errors
    checkCUDAError("memcpy");

    // Part 8: verify the data returned to the host is correct
    for (int i = 0; i < dimA; i++)
    {
        assert(h_a[i] == dimA - 1 - i );
    }

    // free device memory
    hipFree(d_a);
    hipFree(d_b);

    // free host memory
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}
